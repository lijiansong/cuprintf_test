#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cuPrintf.cu"


__global__ void testKernel(int val) {
       cuPrintf("\tValue is:%d\n", val);
}

int main(int argc, char *argv[]) {
       cudaPrintfInit(); 
       testKernel<<< 2, 3 >>>(10); 
       cudaPrintfDisplay(stdout, true); 
       cudaPrintfEnd(); 
       return 0;
}

